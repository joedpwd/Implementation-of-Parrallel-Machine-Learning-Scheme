#include "hip/hip_runtime.h"

#include "kernelSingleBlock.h"

const int Dimensions = 30;
const int Radon = Dimensions + 2;

const int threadsX = 32;
const int threadsY = 32;

__global__ void gaussianKernel(float* devGaussianMat, int height, int width) {

	//Tell one thread to load float devGaussianMat into a shared memory.
	__shared__ float sharedMat[threadsY][threadsX];
	
	int i = 0;
	int j = 0;
	int k = 0;
	int t = 0;
	int x = threadIdx.x;
	int y = threadIdx.y;
	float temp;

	if(y< height && x < width)
		sharedMat[y][x] = *(devGaussianMat + y * width + x);
	__syncthreads();

	//printf("Block Dim %d\t%d", blockDim.x, blockDim.y);

	if (height > blockDim.y || width > blockDim.x)
		printf("Dimensions are too large\n");
	else {

		for (i = 0; i < height; i++) {

			/*if (y == 1 && x == 1) {

				/*for (t = 0; t < width*height; t++) {
					if (t % width == 0)
						printf("\n");
					printf("%.2f\t", *(devGaussianMat + t));
					printf("\n\n");
				}

				for (t = 0; t < width*height; t++) {
					if (t % width == 0)
						printf("\n");
					printf("%.2f\t", sharedMat[t / width][t % width]);
				}
				printf("\n\n");
			}*/


			if (sharedMat[i][i] != 1) {
				temp = sharedMat[i][i];
				if (y == i && x <= width)
					sharedMat[y][x] = sharedMat[y][x] / temp;
				//CHECK FOR 0 ON DIAGONAL
			}
			__syncthreads();

			if (((y < i) || ((y > i) && (y < height))) && x < width)
				sharedMat[y][x] = sharedMat[y][x] - sharedMat[y][i] * sharedMat[i][x];
			__syncthreads();
			/*for (k = 0; k < height; k++) {

				if (k == i)
					continue;
				temp = *(devGaussianMat + k * width + i);
				//printf("%d\n", k*width);
				//printf("temp - %.2f\n", temp);
				for (j = 0; j < width; j++) {
					//printf("k-%d,j-%d val-%.2f\n", k, j, *(devGaussianMat + k * width + j));
					//printf("temp - %.2f\n", temp);
					*(devGaussianMat + k * width + j) = *(devGaussianMat + k * width + j) - (temp * *(devGaussianMat + i * width + j));
					//printf("k-%d,j-%d val-%.2f\n", k,j, *(devGaussianMat + k * width + j));
				}
			}*/
			/*for (t = 0; t < width*height; t++) {
				if (t % width == 0)
					printf("\n");
				printf("%.2f\t", *(devGaussianMat + t));
			}*/
		}

		/*if (y == 1 && x == 1) {
			for (t = 0; t < width*height; t++) {
				if (t % width == 0)
					printf("\n");
				printf("%.2f\t", sharedMat[t / width][t % width]);
			}
			printf("\n\n");
		}*/
	
		if (y < height && x < width)
			//memcpy((devGaussianMat + y * width + x), (sharedMat + y * width + x), sizeof(float));
			*(devGaussianMat + y * width + x) = sharedMat[y][x];

		
	}

	/*if (y == 1 && x == 1) {
		for (t = 0; t < width*height; t++) {
			if (t % width == 0)
				printf("\n");
			printf("%.5f\t", *(devGaussianMat + t));
		}
		printf("\n\n");
	}*/

}
int singleBlock()
{

	//TODO
	//Functionality to read a hypothesis, convert to matrix for guassian elimination. Get Dimensions
	//Allocate memory for 2-D arrays

	int width = Radon;
	int height = Dimensions + 1;
	float r;
	hipEvent_t start, stop;

	float* hostGaussianMat = (float *)malloc(height * width * sizeof(float));

	float* devGaussianMat;

	hipMalloc(&(devGaussianMat), height * width * sizeof(float));

	dim3 grid(1, 1, 1);
	dim3 block(threadsY, threadsX, 1);
	int i = 0;

	//For now we'll just fill matrices with dummy values

	for (i = 0; i < height*width; i++)
		*(hostGaussianMat + i) = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / 10000.0));

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMemcpy(devGaussianMat, hostGaussianMat, height * width * sizeof(float), hipMemcpyHostToDevice);

	gaussianKernel<<<grid, block>>> (devGaussianMat, height, width);

	hipMemcpy(hostGaussianMat, devGaussianMat, height * width * sizeof(float), hipMemcpyDeviceToHost);

	/*for (i = 0; i < width*height; i++) {
		if (i % width == 0)
			printf("\n");
		printf("%.2f\t", *(hostGaussianMat + i));
	}*/

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("Time %.2f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(devGaussianMat);

	free(hostGaussianMat);
	return 0;
}
