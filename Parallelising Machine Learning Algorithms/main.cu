#include "hip/hip_runtime.h"
#include "main.h"

const int print = 1;

//Get the dimensions of the data
	//Set Valuues
	//h will be a hyper parameter passed to the program.

const int d = 2;

const int r = d + 2; //Assuming d = 2

const int h = 3; //Hyper parameter

const int m = d + 1; //Equivalent to d + 1

const int rh = pow(r, h);

int main(int argc, char **argv) {

	//Used for Iteration
	int i=0;
	int j=0;
	int k=0;

	//Perform rh instances of ML problem

	//Size of Data is r^h * d, where d is the no of features
	double *data = (double *)malloc(sizeof(double) * rh * d);


	//Create a vector of threads, one thread per execution of radon machine operation.
	//std::thread *thArray = (std::thread *)malloc(sizeof(std::thread) * pow(r, h - 1));
	std::vector<std::thread> thVect;

	
	//Read data in from CSV, data is stored in long long type and casted back into double type.
	std::ifstream dataFile;
	std::string t;
	std::string::size_type sz;
	dataFile.open("C:/Users/jxd45/Documents/Python Scripts/csvtest.csv");
	long long *test = (long long *)malloc(sizeof(long long));
	if (dataFile.is_open())
	{
		while (std::getline(dataFile, t))
		{
			//std::cout << t << '\n';
			sz = 0;
			for (j = 0; j < d; j++) {
				*test = std::stoll(t.substr(sz), &sz);
				sz++;
				*(data + (i++)) = *reinterpret_cast<double *>(test);
			}
		}
		dataFile.close();
	}
	else
	{
		std::cout << "Unable to open file";

		return 0;
	}
	
	//Check the GPU capabilities



	if (print == 1) {
		for (i = 0; i < d* 4; i++) {
			printf("%.5f\n", *(data + i));
		}
		printf("\n");
	}
	
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	startRadonMachine(data);
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	if (print == 1) {
		for (i = 0; i < d *4; i++) {
			printf("%.5f\n", *(data + i));
		}
		printf("\n");
	}
	auto duration = duration_cast<microseconds>(t2 - t1).count();

	std::cout << duration << " microseconds";
	
	
	free(test);
	free(data);
}

void startRadonMachine(double *dataPoints ) {

	int i, j;
	double *devData;
	double *devEquationData;
	double *devSolvedEquations;
	int *devNofEquation;
	int maxThreads = 4;
	int threads;
	int noOfEquations;
	int equationsPerThread;
	std::vector<std::thread> thVect;

	const dim3 blockSize(16, 16, 1);
	const dim3 gridSize(1, 1, 1);

	hipsolverHandle_t cuSolver = NULL;
	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;

	/* Initialise cuSolver*/
	status = hipsolverDnCreate(&cuSolver);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	//Allocate A and B (A -> (m * m)), (B->1*m)) for r^h instances
	hipMalloc(&devEquationData, (sizeof(double) * m * (m + 1))*(rh/r));
	hipMalloc(&devSolvedEquations, (sizeof(double) * m)*(rh / r));
	hipMalloc(&devData, sizeof(double) * rh * d);
	hipMemcpy(devData, dataPoints, sizeof(double) * rh * d, hipMemcpyHostToDevice);

	hipMalloc(&devNofEquation, sizeof(int));
	

	for (i = 0; i < h; i++) {
		noOfEquations = pow(r, h - 1 - i);
		hipMemcpy(devNofEquation, &noOfEquations, sizeof(int), hipMemcpyHostToDevice);
		configureEquations << < gridSize, blockSize >> > (devData, devEquationData, devNofEquation);

		threads = (noOfEquations > maxThreads ? maxThreads : noOfEquations);
		equationsPerThread = noOfEquations / threads;
		printf("%d threads %d equationsPerThread\n", threads, equationsPerThread);

		for (j = 0; j < threads; j++) {
			thVect.push_back(std::thread(radonInstance, cuSolver, j, (devEquationData + (j*equationsPerThread*m * (m + 1))), equationsPerThread, devSolvedEquations));
			//radonInstance((data + (d*j*r)), d);
		}
		for (std::thread & th : thVect)
		{
			// If thread Object is Joinable then Join that thread.
			if (th.joinable())
				th.join();
		}
		thVect.clear();

		solveEquations << < gridSize, blockSize >> > (devData, devSolvedEquations, devNofEquation);
		//Will sort memory out in thread
		/*for (j = 0; j < pow(r, h-1 - i); j++) {
			for(k=0;k<d;k++)
				*(data + (j*d) + k) = *(data + (r*j*d) + k);
		}*/
	}

	hipMemcpy(dataPoints, devData, sizeof(double) * rh * d, hipMemcpyDeviceToHost);
	
	if (cuSolver) hipsolverDnDestroy(cuSolver);
	/*int i, j;

	const int r = d + 2; //Assuming d = 2

	const int m = d + 1;

	double *hostA = (double *)(malloc(m*m * sizeof(double)));
	double *hostB = (double *)(malloc(m * sizeof(double)));
	double *hostX = (double *)(malloc(m * sizeof(double)));
	double *hostLU = (double *)(malloc(m*m * sizeof(double)));
	int *hostIpiv = (int *)(malloc(m * sizeof(int)));
	int *hostInfo = (int *)(malloc(sizeof(int)));

	long long *Acopy;
	long long *Bcopy;
	long long *Xcopy;

	//Format raw data into correct format for LU factorisation.
	for (j = 0; j < d; j++) {
		hostB[j] = -*(dataPoints+j);
	}

	hostB[j] = -1;

	for (i = 1; i < r; i++) {
		for (j = 0; j < d; j++) {
			hostA[(i - 1)*m + j] = *(dataPoints + (i*d) + j);
		}
		hostA[(i - 1)*m + j] = 1;
	}

	//Perform LU Factorisation
	denseLUSolver(hostA, hostB, hostX, hostLU, hostIpiv, hostInfo, m);*/

	/*for (i = 0; i < m; i++)
		printf("X\t%d\t%.4f\n", i, *(hostX+i));*/

	/*double lambda = 1;
	double *hypothesis = (double *)malloc(sizeof(double)*d);

	//Obtain correct hypothesis for instance. We have chosen to obtain the hypothesis by
	//using the positive index set I. L0 is always 1 (positive) and is therefore always in this set.
	//Therefore the first loop is to copy the first data points for this value into the 
	//hypothesis array. This array is the sum of data points whose index corresponds to the
	//values in the positive index set I.

	for (i = 0; i < d; i++)
		hypothesis[i] = *(dataPoints + i);

	for (i = 0; i < m; i++) {
		if (*(hostX + i) >= 0) {
			lambda += *(hostX + i);
			for (j = 0; j < d; j++)
				hypothesis[j] += *(hostX + i) * *(dataPoints + ((i + 1) *d) + j);
		}
		else
			continue;
	}

	//printf("RESULT\n");

	for (i = 0; i < d; i++) {
		printf("%.5f\n", hypothesis[i]);
	}
	//printf("%.5f\n", lambda);
	for (i = 0; i < d; i++) {
		hypothesis[i] /= lambda;
		//printf("%.5f\n", lambda);
		//printf("%.5f\n", hypothesis[i]);
		*(dataPoints + i) = hypothesis[i];
	}

	Acopy = reinterpret_cast<long long *>(hostA);
	Bcopy = reinterpret_cast<long long *>(hostB);
	Xcopy = reinterpret_cast<long long *>(hostX);

	/*std::ofstream A, B, X;
	A.open("C:/Users/jxd45/Documents/Python Scripts/A.csv", std::ios::out | std::ios::app);
	B.open("C:/Users/jxd45/Documents/Python Scripts/B.csv", std::ios::out | std::ios::app);
	X.open("C:/Users/jxd45/Documents/Python Scripts/X.csv", std::ios::out | std::ios::app);
	if (A.fail() | B.fail() | X.fail()) {
		std::cout << "Unable to open file";
	}
	else {
		for (i = 0; i < m; i++) {
			for (j = 0; j < m-1; j++) {
				A << *(Acopy + i * m + j) << ",";
			}
			A << *(Acopy + i * m + j) << std::endl;
			B << *(Bcopy + i) << std::endl;
			X << *(Xcopy + i) << std::endl;
		}
	}*/

	/*free(hostA);
	free(hostB);
	free(hostX);
	free(hostLU);
	free(hostIpiv);
	free(hostInfo);
	free(hypothesis);*/
}

void radonInstance(hipsolverHandle_t cuSolver, int threadId, double *data, int equations, double *solvedEquations)
{
	/*double *hostA, double *hostB, double *hostX, double *LU, int *Ipiv, int *info, int m*/

	hipStream_t stream = NULL;
	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;	/*Stores Error value for cusolver function calls*/

	/*Used to handle generic cuda errors*/
	hipError_t c1 = hipSuccess;
	hipError_t c2 = hipSuccess;

	double *d_A = NULL; /* device copy of A */
	double *d_B = NULL; /* device copy of B */
	int *d_Ipiv = NULL; /* pivoting sequence */
	int *d_info = NULL; /* error info for cuSolverDn */
	int  lwork = 0;     /* size of workspace for suSolverDn */
	double *d_work = NULL; /* device workspace for getrf, will be allocated using lwork */

	const int lda = m;
	const int ldb = m;

	const int pivot = 1; /*By default we will be using pivoting (pivot = 1)*/

	c1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	assert(hipSuccess == c1);

	c1 = hipMalloc((void**)&d_Ipiv, sizeof(int) * m);
	c2 = hipMalloc((void**)&d_info, sizeof(int));
	assert(hipSuccess == c1);
	assert(hipSuccess == c2);

	mtx.lock();
	status = hipsolverSetStream(cuSolver, stream);
	//c1 = hipDeviceSynchronize();
	//c1 = hipStreamSynchronize(stream);
	//assert(hipSuccess == c1);
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	status = hipsolverDnDgetrf_bufferSize(
		cuSolver,
		m,
		m,
		d_A,
		lda,
		&lwork);
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	//mtx.unlock();

	c1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
	assert(hipSuccess == c1);

	/* Perform LU Factorisation*/

	for (int i = 0; i < equations; i++) {

		d_A = (data + (i*m * (m + 1)));
		d_B = (data + (m*m) + (i*m * (m + 1)));
		//c1 = hipDeviceSynchronize();
		c1 = hipStreamSynchronize(stream);
		assert(hipSuccess == c1);
		//printM<< <1, 1 >> > (m,m, d_A,"A");
		//printM << <1, 1 >> > (m, m, d_B, "B");
		if (pivot) {
			status = hipsolverDnDgetrf(
				cuSolver,
				m,
				m,
				d_A,
				lda,
				d_work,
				d_Ipiv,
				d_info);
		}
		else {
			status = hipsolverDnDgetrf(
				cuSolver,
				m,
				m,
				d_A,
				lda,
				d_work,
				NULL,
				d_info);
		}
		/* Wait until device has finished */
		//c1 = hipDeviceSynchronize();
		c1 = hipStreamSynchronize(stream);
		assert(HIPSOLVER_STATUS_SUCCESS == status);
		assert(hipSuccess == c1);

		if (pivot) {
			status = hipsolverDnDgetrs(
				cuSolver,
				HIPBLAS_OP_N,
				m,
				1, /* nrhs */
				d_A,
				lda,
				d_Ipiv,
				d_B,
				ldb,
				d_info);
		}
		else {
			status = hipsolverDnDgetrs(
				cuSolver,
				HIPBLAS_OP_N,
				m,
				1, /* nrhs */
				d_A,
				lda,
				NULL,
				d_B,
				ldb,
				d_info);
		}
	
		//c1 = hipDeviceSynchronize();
		c1 = hipStreamSynchronize(stream);
		assert(HIPSOLVER_STATUS_SUCCESS == status);
		assert(hipSuccess == c1);
		//c1 = hipMemcpy(hostX, d_B, sizeof(double)*m, hipMemcpyDeviceToHost);

		devMemoryCopy << <1, 1 >> > (d_B, (solvedEquations + (threadId*equations*m) + i * m), m);
	}

	mtx.unlock();

	/* free resources */
	if (d_A) hipFree(d_A);
	if (d_B) hipFree(d_B);
	if (d_Ipiv) hipFree(d_Ipiv);
	if (d_info) hipFree(d_info);
	if (d_work) hipFree(d_work);


	if (stream) hipStreamDestroy(stream);

	//Not
	//hipDeviceReset();

}

/*double lambda = 0;
	double hypothesis[d];

	for (i = 0; i < d; i++)
		hypothesis[i] = 0;

	for (i = 0; i < m; i++) {
		if (*(hostX + i) < 0) {
			lambda += *(hostX + i);
			for (j = 0; j < d; j++)
				hypothesis[j] += *(hostX + i) * data[i + 1][j];
		}
		else
			continue;
	}*/
