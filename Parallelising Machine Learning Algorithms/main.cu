#include "hip/hip_runtime.h"
#include "main.h"

const int print = 1;

//Get the dimensions of the data
//Set Valuues
//h will be a hyper parameter passed to the program.

/*const int d = 12;

const int r = d + 2;

const int h = 5; //Hyper parameter

const int m = d + 1; //Equivalent to d + 1

const int rh = pow(r, h);*/

//"C:\Users\jxd45\Documents\Python Scripts\big.csv"

int main(int argc, char *argv[]) {
	
	/*hipError_t c1;

	size_t *s =  (size_t *)malloc(sizeof(size_t));
	*s = 1000;
	c1 = hipDeviceGetLimit(s, hipLimitPrintfFifoSize);
	assert(hipSuccess == c1);
	c1 = hipDeviceSetLimit(hipLimitPrintfFifoSize, *s * 20);
	assert(hipSuccess == c1);*/

	int d;
	int h;
	std::string inputFile;

	//printf("%d", argc);

	//if (argc == 4) {
		for(int i=0; i < argc; i++)
			std::cout << argv[i] << std::endl;

		inputFile = argv[1];
		d = atoi(argv[2]);
		h = atoi(argv[3]); //Hyper parameter
	//}

	int m = d + 1; //Equivalent to d + 1
	int r = d + 2; //Radon number
	int rh = pow(r, h);

	//Used for Iteration
	int i=0;
	int j=0;

	//Size of Data is r^h * d, where d is the no of features
	double *data = (double *)malloc(sizeof(double) * rh * d);

	//Create a vector of threads, one thread per execution of radon machine operation.
	//std::thread *thArray = (std::thread *)malloc(sizeof(std::thread) * pow(r, h - 1));
	std::vector<std::thread> thVect;

	
	//Read data in from CSV, data is stored in long long type and casted back into double type.
	std::ifstream dataFile;
	std::string t;
	std::string::size_type sz;

	dataFile.open(inputFile); 
	long long *test = (long long *)malloc(sizeof(long long));
	if (dataFile.is_open())
	{
		while (std::getline(dataFile, t))
		{
			//std::cout << t << '\n';
			sz = 0;
			for (j = 0; j < d; j++) {
				t = t.substr(sz);
				*test = std::stoll(t, &sz);
				sz++;
				*(data + (i++)) = *reinterpret_cast<double *>(test);
			}
		}
		dataFile.close();
	}
	else
	{
		std::cout << "Unable to open file";

		return 0;
	}
	
	//Check the GPU capabilities


	//Debugging Purposes
	if (print == 1) {
		for (i = 0; i < d+4; i++) {
			printf("%.5f\n", *(data + i));
		}
		printf("\n");
	}
	
	//Start timer
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	RadonMachineInitialise(d,h,data);
	//End Timers
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	
	if (print == 1) {
		for (i = 0; i < d+4; i++) {
			printf("%.5f\n", *(data + i));
		}
		printf("\n");
	}
	
	auto duration = duration_cast<microseconds>(t2 - t1).count();

	std::cout << duration << " microseconds";
	
	
	free(test);
	free(data);
}

void RadonMachineInitialise(int d, int h, double *dataPoints ) {
	int m = d + 1;
	int r = d + 2;
	int rh = pow(r, h);
	
	int h1 = h;
	int hmax;
	int numInstances;
	int i, j;
	double *subsetDataPoints;
	int subsetSize;

	size_t problemAllocation = sizeof(double) * rh * d;
	size_t devFree;
	size_t devTotal;
	

	hipError_t c1;

	c1 = hipMemGetInfo(&devFree, &devTotal);
	assert(hipSuccess == c1);

	if (devFree > problemAllocation)
		RadonMachineInstance(d, h, dataPoints);
	else {

	
		hmax = getMaxAllocation(devFree, d);
		while (h1 > hmax) {
			numInstances = pow(r, h1 - hmax);

			for (i = 0; i < numInstances; i++) {
				subsetSize = pow(r, hmax) * d * sizeof(double);
				subsetDataPoints = dataPoints + (i * subsetSize);
				RadonMachineInstance(d, hmax, subsetDataPoints);
			}
			for (i = 0; i < numInstances; i++) {
				//Collapse Memory
				for (j = 0; j < d; j++)
					*(dataPoints + (i*d) + j) = *(dataPoints + (i*subsetSize) + j);
			}
			h1 -= hmax;
			hmax = getMaxAllocation(devFree, d);
		}
		RadonMachineInstance(d, h1, dataPoints);
	}
}

int getMaxAllocation(size_t mem, int d) {
	size_t m1 = 0;
	int h = 0;
	int r = d + 2;
	int rh;
	do {
		rh = pow(r, h);
		m1 = sizeof(double) * rh * d;
		h++;
	} while (m1 < mem);
	
	return h - 1;
}

void RadonMachineInstance(int d, int h, double *dataPoints) {

	int m = d + 1;
	int r = d + 2;
	int rh = pow(r, h);

	int i, j;
	double *devData;
	double *devEquationData;
	double *devSolvedEquations;
	double *hypothesisWorkspace;

	int *pivotArray;
	int *infoArray;
	double **Aarrays;

	double *currentData;
	int *currentPiv;
	int *currentInfo;
	double **currentAarr;

	int *devNofEquation;
	int maxThreads = r;
	int threads;
	int noOfEquations;
	int equationsPerThread;
	hipError_t c1;
	std::vector<std::thread> thVect;
	hipStream_t *streams = NULL;

	c1 = hipMalloc(&Aarrays, sizeof(double *) * rh/r * d);
	assert(hipSuccess == c1);
	c1 = hipMalloc(&pivotArray, sizeof(int) * rh/r * d * m);
	assert(hipSuccess == c1);
	c1 = hipMalloc(&infoArray, sizeof(int) * rh/r * d);
	assert(hipSuccess == c1);

	hipblasStatus_t cblsStat;
	hipblasHandle_t *cblsContexts = (hipblasHandle_t *)malloc(maxThreads * sizeof(hipblasHandle_t));

	//Create Streams
	streams = (hipStream_t *)malloc(maxThreads * sizeof(hipStream_t));

	//Allocate size of heap for device
	//c1 = hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(double)*d * 16 * 16 * 8 * 8);
	//assert(hipSuccess == c1);
	//cudaThreadSetLimit(hipLimitMallocHeapSize, sizeof(double)*d*16*16*8*8);

	for (i = 0; i < maxThreads; i++) {
		c1 = hipStreamCreateWithFlags(streams + i, hipStreamNonBlocking);
		assert(hipSuccess == c1);
		cblsStat = hipblasCreate(cblsContexts + i);
		assert(cblsStat == HIPBLAS_STATUS_SUCCESS);
	}

	const dim3 blockSize(16, 16, 1);
	const dim3 gridSize(8, 8, 1);

	//Allocate space for Equations, solved equations and space for data on the device. Then copy data to device.
	//Allocate A and B (A -> (m * m)), (B->1*m)) for r^h instances
	c1 = hipMalloc(&devEquationData, (sizeof(double) * m * (m + 1))*(rh / r));
	assert(hipSuccess == c1);
	c1 = hipMalloc(&devSolvedEquations, (sizeof(double) * m)*(rh / r));
	assert(hipSuccess == c1);
	c1 = hipMalloc(&hypothesisWorkspace, (sizeof(double) * d)*(rh / r));
	assert(hipSuccess == c1);
	c1 = hipMalloc(&devData, sizeof(double) * rh * d);
	assert(hipSuccess == c1);
	c1 = hipMemcpy(devData, dataPoints, sizeof(double) * rh * d, hipMemcpyHostToDevice);
	assert(hipSuccess == c1);
	//Maintains the number of equations to be solved at each level of the radon tree
	c1 = hipMalloc(&devNofEquation, sizeof(int));
	assert(hipSuccess == c1);
	//printM << <1, 1 >> > (m, m, devData, "A");

	initAarr << < gridSize, blockSize >> > (d, Aarrays, devEquationData, rh/r);

	for (i = 0; i < h; i++) {
		noOfEquations = pow(r, h - 1 - i);
		hipMemcpy(devNofEquation, &noOfEquations, sizeof(int), hipMemcpyHostToDevice);
		configureEquations << < gridSize, blockSize >> > (d, devData, devEquationData, devNofEquation);
		hipDeviceSynchronize();
		threads = (noOfEquations > maxThreads ? maxThreads : noOfEquations);
		equationsPerThread = noOfEquations / threads;

		//printf("%d threads %d equationsPerThread\n", threads, equationsPerThread);
		hipDeviceSynchronize();
		//printM << <1, 1, 0 >> > (pow(r, h - i)*d, 1, devData, "A");
		hipDeviceSynchronize();
		for (j = 0; j < threads; j++) {
			//currentData = (devEquationData + (j*equationsPerThread*m * (m + 1)));
			currentInfo = infoArray + j * equationsPerThread;
			currentAarr = Aarrays + j * equationsPerThread;
			currentPiv = pivotArray + (j * m * equationsPerThread);
			thVect.push_back(std::thread(radonInstance, d, cblsContexts + j, j, currentAarr, currentPiv, currentInfo, equationsPerThread, devSolvedEquations, streams + j));
		}
		for (std::thread & th : thVect)
		{
			if (th.joinable())
				th.join();
		}
		thVect.clear();
		solveEquations << < gridSize, blockSize >> > (d, devData, devSolvedEquations, devNofEquation, hypothesisWorkspace);
		hipDeviceSynchronize();
		//printM << <1, 1, 0 >> > (pow(r, h - i), 1, devData, "A");
	}

	hipMemcpy(dataPoints, devData, sizeof(double) * rh * d, hipMemcpyDeviceToHost);

	for (i = 0; i < maxThreads; i++) {
		cblsStat = hipblasDestroy(*(cblsContexts + i));
		assert(cblsStat == HIPBLAS_STATUS_SUCCESS);
		c1 = hipStreamDestroy(*(streams + i));
		assert(hipSuccess == c1);
	}

	c1 = hipFree(devData);
	assert(hipSuccess == c1);
	c1 = hipFree(devSolvedEquations);
	assert(hipSuccess == c1);
	c1 = hipFree(devEquationData);
	assert(hipSuccess == c1);
	c1 = hipFree(hypothesisWorkspace);
	assert(hipSuccess == c1);
	c1 = hipFree(streams);
	assert(hipSuccess == c1);
	c1 = hipFree(infoArray);
	assert(hipSuccess == c1);
	c1 = hipFree(pivotArray);
	assert(hipSuccess == c1);

	free(Aarrays);
}

void radonInstance(int d, hipblasHandle_t *cublas, int threadId, double **data, int *piv, int *info, int equations, double *solvedEquations, hipStream_t *s)
{
	
	int m = d + 1;
	hipblasStatus_t cblsStat;

	cblsStat = hipblasSetStream(*cublas, *s);
	assert(cblsStat == HIPBLAS_STATUS_SUCCESS);

	/*Used to handle generic cuda errors*/
	hipError_t c1 = hipSuccess;
	hipError_t c2 = hipSuccess;

	double *d_A = NULL; /* device copy of A */
	double *d_B = NULL; /* device copy of B */
	int *d_Ipiv = NULL; /* pivoting sequence */
	int *d_info = NULL; /* error info for cuSolverDn */
	int  lwork = 0;     /* size of workspace for suSolverDn */
	double *d_work = NULL; /* device workspace for getrf, will be allocated using lwork */

	const int lda = m;
	const int ldb = m;

	const int pivot = 1; /*By default we will be using pivoting (pivot = 1)*/
	mtx.lock();
	c1 = hipMalloc((void**)&d_Ipiv, sizeof(int) * m);
	c2 = hipMalloc((void**)&d_info, sizeof(int));
	assert(hipSuccess == c1);
	assert(hipSuccess == c2);

	hipblasDgetrfBatched(*cublas, m, data, m, piv, info, equations);

	hipStreamSynchronize(*s);
	devMemoryCopy << <1, 1, 0, *s >> > (m, d_B, (solvedEquations + (threadId*equations*m) + i * m), m);
	hipStreamSynchronize(*s);

	/* free resources */
	if (d_Ipiv) hipFree(d_Ipiv);
	if (d_info) hipFree(d_info);
	if (d_work) hipFree(d_work);
	mtx.unlock();
}