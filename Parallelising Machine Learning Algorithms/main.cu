#include "hip/hip_runtime.h"
#include "main.h"

const int print = 1;

//Get the dimensions of the data
//Set Valuues
//h will be a hyper parameter passed to the program.

/*const int d = 12;

const int r = d + 2;

const int h = 5; //Hyper parameter

const int m = d + 1; //Equivalent to d + 1

const int rh = pow(r, h);*/

//"C:\Users\jxd45\Documents\Python Scripts\big.csv"

int main(int argc, char *argv[]) {
	
	/*hipError_t c1;

	size_t *s =  (size_t *)malloc(sizeof(size_t));
	*s = 1000;
	c1 = hipDeviceGetLimit(s, hipLimitPrintfFifoSize);
	assert(hipSuccess == c1);
	c1 = hipDeviceSetLimit(hipLimitPrintfFifoSize, *s * 20);
	assert(hipSuccess == c1);*/

	int d;
	int h;
	std::string inputFile;

	//printf("%d", argc);

	//if (argc == 4) {
		for(int i=0; i < argc; i++)
			std::cout << argv[i] << std::endl;

		inputFile = argv[1];
		d = atoi(argv[2]);
		h = atoi(argv[3]); //Hyper parameter
	//}

	int m = d + 1; //Equivalent to d + 1
	int r = d + 2; //Radon number
	int rh = pow(r, h);

	//Used for Iteration
	int i=0;
	int j=0;

	//Size of Data is r^h * d, where d is the no of features
	double *data = (double *)malloc(sizeof(double) * rh * d);

	//Create a vector of threads, one thread per execution of radon machine operation.
	//std::thread *thArray = (std::thread *)malloc(sizeof(std::thread) * pow(r, h - 1));
	std::vector<std::thread> thVect;

	
	//Read data in from CSV, data is stored in long long type and casted back into double type.
	std::ifstream dataFile;
	std::string t;
	std::string::size_type sz;

	dataFile.open(inputFile); 
	long long *test = (long long *)malloc(sizeof(long long));
	if (dataFile.is_open())
	{
		while (std::getline(dataFile, t))
		{
			//std::cout << t << '\n';
			sz = 0;
			for (j = 0; j < d; j++) {
				t = t.substr(sz);
				*test = std::stoll(t, &sz);
				sz++;
				*(data + (i++)) = *reinterpret_cast<double *>(test);
			}
		}
		dataFile.close();
	}
	else
	{
		std::cout << "Unable to open file";

		return 0;
	}
	
	//Check the GPU capabilities


	//Debugging Purposes
	if (print == 1) {
		for (i = 0; i < d+4; i++) {
			printf("%.5f\n", *(data + i));
		}
		printf("\n");
	}
	
	//Start timer
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	startRadonMachine(d,h,data);
	//End Timers
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	
	if (print == 1) {
		for (i = 0; i < d+4; i++) {
			printf("%.5f\n", *(data + i));
		}
		printf("\n");
	}
	
	auto duration = duration_cast<microseconds>(t2 - t1).count();

	std::cout << duration << " microseconds";
	
	
	free(test);
	free(data);
}

void startRadonMachine(int d, int h, double *dataPoints ) {

	int m = d + 1;
	int r = d + 2;
	int rh = pow(r, h);

	int i, j;
	double *devData;
	double *devEquationData;
	double *devSolvedEquations;
	double *hypothesisWorkspace;
	int *devNofEquation;
	int maxThreads = r;
	int threads;
	int noOfEquations;
	int equationsPerThread;
	hipError_t c1;
	std::vector<std::thread> thVect;
	hipStream_t *streams = NULL;

	//Create Streams
	streams = (hipStream_t *)malloc(maxThreads * sizeof(hipStream_t));
	
	//Allocate size of heap for device
	//c1 = hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(double)*d * 16 * 16 * 8 * 8);
	//assert(hipSuccess == c1);
	//cudaThreadSetLimit(hipLimitMallocHeapSize, sizeof(double)*d*16*16*8*8);

	for (i = 0; i < maxThreads; i++) {
		c1 = hipStreamCreateWithFlags(streams+i, hipStreamNonBlocking);
		assert(hipSuccess == c1);
	}

	const dim3 blockSize(16, 16, 1);
	const dim3 gridSize(8, 8, 1);

	hipsolverHandle_t cuSolver = NULL;
	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;

	/* Initialise cuSolver*/
	status = hipsolverDnCreate(&cuSolver);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	//Allocate space for Equations, solved equations and space for data on the device. Then copy data to device.
	//Allocate A and B (A -> (m * m)), (B->1*m)) for r^h instances
	c1 = hipMalloc(&devEquationData, (sizeof(double) * m * (m + 1))*(rh/r));
	assert(hipSuccess == c1);
	c1 = hipMalloc(&devSolvedEquations, (sizeof(double) * m)*(rh / r));
	assert(hipSuccess == c1);
	c1 = hipMalloc(&hypothesisWorkspace, (sizeof(double) * d)*(rh / r));
	assert(hipSuccess == c1);
	c1 = hipMalloc(&devData, sizeof(double) * rh * d);
	assert(hipSuccess == c1);
	c1 = hipMemcpy(devData, dataPoints, sizeof(double) * rh * d, hipMemcpyHostToDevice);
	assert(hipSuccess == c1);
	//Maintains the number of equations to be solved at each level of the radon tree
	c1 = hipMalloc(&devNofEquation, sizeof(int));
	assert(hipSuccess == c1);
	//printM << <1, 1 >> > (m, m, devData, "A");

	for (i = 0; i < h; i++) {
		noOfEquations = pow(r, h - 1 - i);
		hipMemcpy(devNofEquation, &noOfEquations, sizeof(int), hipMemcpyHostToDevice);
		configureEquations << < gridSize, blockSize >> > (d, devData, devEquationData, devNofEquation);
		hipDeviceSynchronize();
		threads = (noOfEquations > maxThreads ? maxThreads : noOfEquations);
		equationsPerThread = noOfEquations / threads;
		
		//printf("%d threads %d equationsPerThread\n", threads, equationsPerThread);
		hipDeviceSynchronize();
		//printM << <1, 1, 0 >> > (pow(r, h - i)*d, 1, devData, "A");
		hipDeviceSynchronize();
		for (j = 0; j < threads; j++) {
			thVect.push_back(std::thread(radonInstance,d, cuSolver, j, (devEquationData + (j*equationsPerThread*m * (m + 1))), equationsPerThread, devSolvedEquations, streams + j));
		}
		for (std::thread & th : thVect)
		{
			if (th.joinable())
				th.join();
		}
		thVect.clear();
		solveEquations << < gridSize, blockSize >> > (d, devData, devSolvedEquations, devNofEquation, hypothesisWorkspace);
		hipDeviceSynchronize();
		//printM << <1, 1, 0 >> > (pow(r, h - i), 1, devData, "A");
	}

	hipMemcpy(dataPoints, devData, sizeof(double) * rh * d, hipMemcpyDeviceToHost);
	
	if (cuSolver) hipsolverDnDestroy(cuSolver);

	for (i = 0; i < maxThreads; i++) {
		c1 = hipStreamDestroy(*(streams + i));
		assert(hipSuccess == c1);
	}
	
	c1 = hipFree(devData);
	assert(hipSuccess == c1);
	c1 = hipFree(devSolvedEquations);
	assert(hipSuccess == c1);
	c1 = hipFree(devEquationData);
	assert(hipSuccess == c1);
	c1 = hipFree(hypothesisWorkspace);
	assert(hipSuccess == c1);

	free(streams);

}

void radonInstance(int d, hipsolverHandle_t cuSolver, int threadId, double *data, int equations, double *solvedEquations, hipStream_t *s)
{
	mtx.lock();
	int m = d + 1;
	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;	/*Stores Error value for cusolver function calls*/

	/*Used to handle generic cuda errors*/
	hipError_t c1 = hipSuccess;
	hipError_t c2 = hipSuccess;

	double *d_A = NULL; /* device copy of A */
	double *d_B = NULL; /* device copy of B */
	int *d_Ipiv = NULL; /* pivoting sequence */
	int *d_info = NULL; /* error info for cuSolverDn */
	int  lwork = 0;     /* size of workspace for suSolverDn */
	double *d_work = NULL; /* device workspace for getrf, will be allocated using lwork */

	const int lda = m;
	const int ldb = m;

	const int pivot = 1; /*By default we will be using pivoting (pivot = 1)*/

	c1 = hipMalloc((void**)&d_Ipiv, sizeof(int) * m);
	c2 = hipMalloc((void**)&d_info, sizeof(int));
	assert(hipSuccess == c1);
	assert(hipSuccess == c2);
	
	
	status = hipsolverSetStream(cuSolver, *s);
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	status = hipsolverDnDgetrf_bufferSize(
		cuSolver,
		m,
		m,
		d_A,
		lda,
		&lwork);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	c1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
	assert(hipSuccess == c1);

	/* Perform LU Factorisation*/

	for (int i = 0; i < equations; i++) {

		d_A = (data + (i*m * (m + 1)));
		d_B = (data + (m*m) + (i*m * (m + 1)));
		/*if (threadId == 7 && i == 0) {
			printM << <1, 1, 0, *s >> > (m, m, d_A, "A");
			printf("\n");
			printM << <1, 1, 0, *s >> > (m, 1, d_B, "B");
			printf("\n");
			hipStreamSynchronize(*s);
			//printf("%d\n", lwork);
		}*/
		hipStreamSynchronize(*s);
		if (pivot) {
			status = hipsolverDnDgetrf(
				cuSolver,
				m,
				m,
				d_A,
				lda,
				d_work,
				d_Ipiv,
				d_info);
		}
		else {
			status = hipsolverDnDgetrf(
				cuSolver,
				m,
				m,
				d_A,
				lda,
				d_work,
				NULL,
				d_info);
		}
		
		assert(HIPSOLVER_STATUS_SUCCESS == status);
		hipStreamSynchronize(*s);
		if (pivot) {
			status = hipsolverDnDgetrs(
				cuSolver,
				HIPBLAS_OP_N,
				m,
				1, /* nrhs */
				d_A,
				lda,
				d_Ipiv,
				d_B,
				ldb,
				d_info);
		}
		else {
			status = hipsolverDnDgetrs(
				cuSolver,
				HIPBLAS_OP_N,
				m,
				1, /* nrhs */
				d_A,
				lda,
				NULL,
				d_B,
				ldb,
				d_info);
		}
	
		assert(HIPSOLVER_STATUS_SUCCESS == status);

		//hipStreamSynchronize(*s);
		/*if (threadId == 7 && i == 0) {
			printM << <1, 1, 0, *s >> > (m, 1, d_B, "B");
		}*/
		

		hipStreamSynchronize(*s);
		devMemoryCopy << <1, 1, 0, *s >> > (m, d_B, (solvedEquations + (threadId*equations*m) + i * m), m);
		hipStreamSynchronize(*s);
	}


	/* free resources */
	if (d_Ipiv) hipFree(d_Ipiv);
	if (d_info) hipFree(d_info);
	if (d_work) hipFree(d_work);
	mtx.unlock();
}