#include "denseLUSolverExample.h"

//Function printMatrix is taken from:
//https://docs.nvidia.com/cuda/cusolver/index.html#lu_examples
//Function denseLUSolver has been adapted from the example from
// the same resource.

const int m = 3;
const int lda = m;
const int ldb = m;

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			double Areg = A[row + col * lda];
			printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
		}
	}
}

int denseLUSolver(double *hostA, double *hostB, double *hostX, double *LU, int *Ipiv, int *info, int m)
{
	hipsolverHandle_t cuSolver = NULL;	/*Will be passed to function that will initialise library and allocate resources*/
	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;	/*Stores Error value for cusolver function calls*/
	
	/*Used to handle generic cuda errors*/
	hipError_t c1 = hipSuccess;
	hipError_t c2 = hipSuccess;
	hipError_t c3 = hipSuccess;
	hipError_t c4 = hipSuccess;

	double *d_A = NULL; /* device copy of A */
	double *d_B = NULL; /* device copy of B */
	int *d_Ipiv = NULL; /* pivoting sequence */
	int *d_info = NULL; /* error info for cuSolverDn */
	int  lwork = 0;     /* size of workspace for suSolverDn */
	double *d_work = NULL; /* device workspace for getrf, will be allocated using lwork */

	const int lda = m;
	const int ldb = m;

	const int pivot = 1; /*By default we will be using pivoting (pivot = 1)*/
	const int print = 1; /*Print useful information, (taken from example, see above)*/
	
	if (print)
	{
		printf("example of getrf \n");

		if (pivot) {
			printf("pivot is on : compute P*A = L*U \n");
		}
		else {
			printf("pivot is off: compute A = L*U (not numerically stable)\n");
		}

		printf("A = (matlab base-1)\n");
		printMatrix(m, m, hostA, lda, "A");
		printf("=====\n");

		printf("B = (matlab base-1)\n");
		printMatrix(m, 1, hostB, ldb, "B");
		printf("=====\n");
	}

	/* Initialise cuSolver*/
	status = hipsolverDnCreate(&cuSolver);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	/*Allocate resources on device and copy A and B to device*/
	c1 = hipMalloc((void**)&d_A, sizeof(double) * lda * m);
	c2 = hipMalloc((void**)&d_B, sizeof(double) * m);
	c3 = hipMalloc((void**)&d_Ipiv, sizeof(int) * m);
	c4 = hipMalloc((void**)&d_info, sizeof(int));
	assert(hipSuccess == c1);
	assert(hipSuccess == c2);
	assert(hipSuccess == c3);
	assert(hipSuccess == c4);

	c1 = hipMemcpy(d_A, hostA, sizeof(double)*lda*m, hipMemcpyHostToDevice);
	c2 = hipMemcpy(d_B, hostB, sizeof(double)*m, hipMemcpyHostToDevice);
	assert(hipSuccess == c1);
	assert(hipSuccess == c2);

	/*Get the sizw of the workspace required and store it in lwork.
	Then allocate the workspace and store reference at dwork*/

	status = hipsolverDnDgetrf_bufferSize(
		cuSolver,
		m,
		m,
		d_A,
		lda,
		&lwork);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	c1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
	assert(hipSuccess == c1);

	/* Perform LU Factorisation*/

	if (pivot) {
		status = hipsolverDnDgetrf(
			cuSolver,
			m,
			m,
			d_A,
			lda,
			d_work,
			d_Ipiv,
			d_info);
	}
	else {
		status = hipsolverDnDgetrf(
			cuSolver,
			m,
			m,
			d_A,
			lda,
			d_work,
			NULL,
			d_info);
	}

	/* Wait until device has finished */
	c1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	assert(hipSuccess == c1);

	/* Copy pivot values back to device if pivot is on. Also copy LU array and matrix back to device */
	if (pivot) {
		c1 = hipMemcpy(Ipiv, d_Ipiv, sizeof(int)*m, hipMemcpyDeviceToHost);
	}
	c2 = hipMemcpy(LU, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
	c3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == c1);
	assert(hipSuccess == c2);
	assert(hipSuccess == c3);

	/*Print values according to operation success*/
	if (0 > info) {
		printf("%d-th parameter is wrong \n", info);
		exit(1);
	}
	if (print) {
		if (pivot) {
			printf("pivoting sequence, matlab base-1\n");
			for (int j = 0; j < m; j++) {
				printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
			}
		}
	}
	printf("L and U = (matlab base-1)\n");
	printMatrix(m, m, LU, lda, "LU");
	printf("=====\n");

	/* Using LU decomposition solve for x*/
	if (pivot) {
		status = hipsolverDnDgetrs(
			cuSolver,
			HIPBLAS_OP_N,
			m,
			1, /* nrhs */
			d_A,
			lda,
			d_Ipiv,
			d_B,
			ldb,
			d_info);
	}
	else {
		status = hipsolverDnDgetrs(
			cuSolver,
			HIPBLAS_OP_N,
			m,
			1, /* nrhs */
			d_A,
			lda,
			NULL,
			d_B,
			ldb,
			d_info);
	}
	c1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	assert(hipSuccess == c1);

	c1 = hipMemcpy(hostX, d_B, sizeof(double)*m, hipMemcpyDeviceToHost);
	assert(hipSuccess == c1);

	printf("X = (matlab base-1)\n");
	printMatrix(m, 1, hostX, ldb, "X");
	printf("=====\n");

	printf("%d\n", info);

	/* free resources */
	if (d_A) hipFree(d_A);
	if (d_B) hipFree(d_B);
	if (d_Ipiv) hipFree(d_Ipiv);
	if (d_info) hipFree(d_info);
	if (d_work) hipFree(d_work);

	if (cuSolver) hipsolverDnDestroy(cuSolver);

	hipDeviceReset();
}